#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <torch/torch.h>
#include "simulators/kernels/common.h"

namespace {
    const int ntpb = 512;
    template<typename T, size_t N>
    using TensorRestrict64 = torch::GenericPackedTensorAccessor<T, N, torch::RestrictPtrTraits, int64_t>;
}

template<typename T>
__global__ void _curand_init_states(T *curand_states, long n){
    const long tidx = blockIdx.x * blockDim.x + threadIdx.x;
    if(tidx < n) hiprand_init(0, tidx, 0, &curand_states[tidx]);
}

template<typename T>
void curand_init_states(T *curand_states, long n, hipStream_t stream){
    _curand_init_states<<<(n+ntpb-1)/ntpb, ntpb, 0, stream>>>(curand_states, n);
}

template void curand_init_states<hiprandStateMRG32k3a_t>(hiprandStateMRG32k3a_t*, long, hipStream_t);
template void curand_init_states<hiprandStatePhilox4_32_10_t>(hiprandStatePhilox4_32_10_t*, long, hipStream_t);
